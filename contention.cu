#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>


#define FLTSIZE sizeof(float)

inline int BLK(int data, int blocksize)
{
	return (data + blocksize - 1) / blocksize;
}

__global__ void kernel_vectorAdd (const float* __restrict__ a_d, 
		const float* __restrict__ b_d,
		const int N,
		const int offset,
		float *c_d)
{
	int tid = threadIdx.x + __mul24(blockIdx.x, blockDim.x);

	if(tid < N) {
		c_d[tid + offset] = a_d[tid + offset] + b_d[tid + offset];	
	}
}

int main( int argc, char **argv)
{
	int devid = 0 ;

	int num_streams = 8;

	int N = 1 << 20;

	if(argc >= 2)
		num_streams = atoi(argv[1]);

	if(argc >= 3)
		devid = atoi(argv[2]);

	if(argc >= 4)
		N = atoi(argv[3]);

	hipSetDevice(devid);
/*
	printf("\nrunning %d cuda streams on device %d\n", num_streams, devid);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devid);
	printf("Device Number: %d\n", devid);
	printf("  Device name: %s\n", prop.name);
	printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
	printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
	printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	printf("  Concurrent copy and execution: %s\n",  (prop.deviceOverlap ? "Yes" : "No"));
	printf("  Concurrent kernels: %d\n",  (prop.concurrentKernels));
	printf("  Concurrent copy and kernel execution: %s with %d copy engine(s)\n", 
			(prop.deviceOverlap ? "Yes" : "No"), prop.asyncEngineCount);
*/

	// allocate streams
    hipStream_t *streams = (hipStream_t *) malloc(num_streams * sizeof(hipStream_t));

	// init
    for (int i = 0; i < num_streams; i++) {
        checkCudaErrors(hipStreamCreate(&(streams[i])));
    }

	//------------------------------------------------------------------------//
	// allocate data on the host
	//------------------------------------------------------------------------//
	size_t databytes = N  * FLTSIZE; 

	//float *a_h = (float*) malloc ( N * num_streams * FLTSIZE);
	//float *b_h = (float*) malloc ( N * num_streams * FLTSIZE);
	//float *c_h = (float*) malloc ( N * num_streams * FLTSIZE);

	float *a_h = NULL;
    checkCudaErrors(hipHostMalloc((void **)&a_h, N * num_streams * FLTSIZE));

	float *b_h = NULL;
    checkCudaErrors(hipHostMalloc((void **)&b_h, N * num_streams * FLTSIZE));

	float *c_h = NULL;
    checkCudaErrors(hipHostMalloc((void **)&c_h, N * num_streams * FLTSIZE));

	for(int i=0; i< N * num_streams; i++) {
		a_h[i] = 1.1f;	
		b_h[i] = 2.2f;	
	}

	//------------------------------------------------------------------------//
	// allocate data on the device 
	//------------------------------------------------------------------------//
	float *a_d;
	float *b_d;
	float *c_d;
	hipMalloc((void**)&a_d, N * num_streams * FLTSIZE);
	hipMalloc((void**)&b_d, N * num_streams * FLTSIZE);
	hipMalloc((void**)&c_d, N * num_streams * FLTSIZE);

	// kernel configuration
	dim3 threads = dim3(256, 1, 1);
	dim3 blocks  = dim3(BLK(N, threads.x), 1, 1);

	// create cuda event handles
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	hipEventRecord(start,0);

	// copy data to deivce
	for (int i = 0; i < num_streams; i++) {
		int offset = i * N;
		hipMemcpyAsync(&a_d[offset], &a_h[offset],  databytes, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(&b_d[offset], &b_h[offset],  databytes, hipMemcpyHostToDevice, streams[i]);
	}

	// launch one worker kernel per stream
	for (int i = 0; i < num_streams; i++) {
		int offset = i * N;
		kernel_vectorAdd <<< blocks, threads, 0, streams[i] >>> (a_d, 
				                                                 b_d, 
																 N, 
																 offset,
																 c_d);
	}

	// copy data back to host
	for (int i = 0; i < num_streams; i++) {
		int offset = i * N;
		hipMemcpyAsync(&c_h[offset], &c_d[offset],  databytes, hipMemcpyDeviceToHost, streams[i]);
	}

	// required for async copy
	//hipDeviceSynchronize();

	//hipEventSynchronize(stop);
    hipEventRecord(stop, 0);

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter=0;
    while (hipEventQuery(stop) == hipErrorNotReady)
    {
        counter++;
    }


	
	float gpuTime_ms= 0;
	hipEventElapsedTime(&gpuTime_ms, start, stop);

	//printf("runtime (ms) : %f\n", gpuTime_ms);

	/*
	// check data
	bool success = 1;
	for(int i=0; i< N * num_streams; i++) {
		if (abs(c_h[i] - 3.3f) > 1e-6) {
			fprintf(stderr, "%d : %f  (error)!\n", i, c_h[i]);
			success = 0;
			break;
		}
	}

	if(success) {
		printf("\nSuccess! Exit.\n");	
	}
	*/

	//------------------------------------------------------------------------//
	// free 
	//------------------------------------------------------------------------//
    for (int i = 0; i < num_streams; i++) {
        checkCudaErrors(hipStreamDestroy(streams[i]));
    }

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

	hipHostFree(a_h);
	hipHostFree(b_h);
	hipHostFree(c_h);

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

	hipDeviceReset();

	return 0;
}
